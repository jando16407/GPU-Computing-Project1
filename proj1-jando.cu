/* ==================================================================
	The basic SDH algorithm implementation for 3D data
	To compile: nvcc spacial_distance_histogram.c -o SDH in the c4cuda machines
   ==================================================================
*/

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>


#define BOX_SIZE	23000 /* size of the data box on one dimension            */

/* descriptors for single atom in the tree */
typedef struct atomdesc {
	double x_pos;
	double y_pos;
	double z_pos;
} atom;

typedef struct hist_entry{
	//float min;
	//float max;
	long long d_cnt;   /* need a long long type as the count might be huge */
} bucket;


bucket * histogram;		/* list of all buckets in the histogram   */
bucket * histogram_GPU;	/* list of all buckets in the histogram used for GPU computing */
long long total_num_data_points;   /* total number of data points            */
int num_buckets;		/* total number of buckets in the histogram */
double bucket_width;	/* value of w                             */
atom * atom_list;		/* list of all data points                */

/* These are for an old way of tracking time */
struct timezone Idunno;	
struct timeval startTime, endTime;


/* 
	distance of two points in the atom_list 
*/
double p2p_distance(int ind1, int ind2) {
	
	double x1 = atom_list[ind1].x_pos;
	double x2 = atom_list[ind2].x_pos;
	double y1 = atom_list[ind1].y_pos;
	double y2 = atom_list[ind2].y_pos;
	double z1 = atom_list[ind1].z_pos;
	double z2 = atom_list[ind2].z_pos;
		
	return sqrt((x1 - x2)*(x1-x2) + (y1 - y2)*(y1 - y2) + (z1 - z2)*(z1 - z2));
}


/* 
	brute-force SDH solution in a single CPU thread 
*/
int PDH_baseline() {
	int i, j, h_pos;
	double dist;
	
	for(i = 0; i < total_num_data_points; i++) {
		for(j = i+1; j < total_num_data_points; j++) {
			dist = p2p_distance(i,j);
			h_pos = (int) (dist / bucket_width);
			histogram[h_pos].d_cnt++;
		} 
	}
	return 0;
}

/* 
	Kernel version of SDH solution
*/
__global__
void PDH_Kernel(bucket * hist, atom * at_list, long long total_points, double b_width){
	/* Declare variables */
	//Get index of histogram
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j, h_pos;
	double dist;

	/* Compute distance and update histogram */
	for( j=i+1; j<total_points; ++j ){
		double x1 = at_list[i].x_pos;
        double x2 = at_list[j].x_pos;
        double y1 = at_list[i].y_pos;
        double y2 = at_list[j].y_pos;
        double z1 = at_list[i].z_pos;
		double z2 = at_list[j].z_pos;
		
		dist = sqrt((x1 - x2)*(x1-x2) + (y1 - y2)*(y1 - y2) + (z1 - z2)*(z1 - z2));
		h_pos = (int) (dist / b_width);
		hist[h_pos].d_cnt++;	
	}
	//Synchronize all threads
	__syncthreads();
}

/* 
	set a checkpoint and show the (natural) running time in seconds 
*/
double report_running_time_CPU() {
	long sec_diff, usec_diff;
	gettimeofday(&endTime, &Idunno);
	sec_diff = endTime.tv_sec - startTime.tv_sec;
	usec_diff= endTime.tv_usec-startTime.tv_usec;
	if(usec_diff < 0) {
		sec_diff --;
		usec_diff += 1000000;
	}
	printf("\nRunning time for CPU version: %ld.%06ld\n", sec_diff, usec_diff);
	return (double)(sec_diff*1.0 + usec_diff/1000000.0);
}

/*
	Measure time for GPU comptation
*/
double report_running_time_GPU() {
	long sec_diff, usec_diff;
	gettimeofday(&endTime, &Idunno);
	sec_diff = endTime.tv_sec - startTime.tv_sec;
	usec_diff= endTime.tv_usec-startTime.tv_usec;
	if(usec_diff < 0) {
		sec_diff --;
		usec_diff += 1000000;
	}
	printf("\nRunning time for GPU version: %ld.%06ld\n", sec_diff, usec_diff);
	return (double)(sec_diff*1.0 + usec_diff/1000000.0);
}


/* 
	print the counts in all buckets of the histogram 
*/
void output_histogram(){
	int i; 
	long long total_cnt = 0;
	printf("\nHistogram of CPU computing");
	for(i=0; i< num_buckets; i++) {
		if(i%5 == 0) /* we print 5 buckets in a row */
			printf("\n%02d: ", i);
		printf("%15lld ", histogram[i].d_cnt);
		total_cnt += histogram[i].d_cnt;
	  	/* we also want to make sure the total distance count is correct */
		if(i == num_buckets - 1)	
			printf("\n T:%lld \n", total_cnt);
		else printf("| ");
	}
}

/* 
	Prints out the counts in histogram for GPU
*/
void output_histogram_GPU(){
	int i; 
	long long total_cnt = 0;
	printf("\nHistogram of GPU computing");
	for(i=0; i< num_buckets; i++) {
		if(i%5 == 0) /* we print 5 buckets in a row */
			printf("\n%02d: ", i);
		printf("%15lld ", histogram_GPU[i].d_cnt);
		total_cnt += histogram_GPU[i].d_cnt;
	  	/* we also want to make sure the total distance count is correct */
		if(i == num_buckets - 1)	
			printf("\n T:%lld \n", total_cnt);
		else printf("| ");
	}
}

/*
	print the difference in each histogram
*/
void output_histogram_difference(){
	int i; 
	long long total_cnt = 0;
	printf("\nHistogram dirreference of CPU and GPU.");
	for(i=0; i< num_buckets; i++) {
		if(i%5 == 0) /* we print 5 buckets in a row */
			printf("\n%02d: ", i);
		printf("%15lld ", histogram_GPU[i].d_cnt - histogram[i].d_cnt);
		total_cnt += histogram_GPU[i].d_cnt - histogram[i].d_cnt;
	  	/* we also want to make sure the total distance count is correct */
		if(i == num_buckets - 1)	
			printf("\n T:%lld \n", total_cnt);
		else printf("| ");
	}
}


int main(int argc, char **argv)
{
	/* Read command line value */
	total_num_data_points = atoi(argv[1]);
	bucket_width	 = atof(argv[2]);

	/* Initialize variables */
	num_buckets = (int)(BOX_SIZE * 1.732 / bucket_width) + 1;
	histogram = (bucket *)malloc(sizeof(bucket)*num_buckets);
	histogram_GPU = (bucket *)malloc(sizeof(bucket)*num_buckets);
	atom_list = (atom *)malloc(sizeof(atom)*total_num_data_points);

	/* Declare variables */
	int i;
	bucket * d_histogram_GPU;
	atom * d_atom_list;

	
	/* generate data following a uniform distribution */
	srand(1);
	for(i = 0;  i < total_num_data_points; i++) {
		atom_list[i].x_pos = ((double)(rand()) / RAND_MAX) * BOX_SIZE;
		atom_list[i].y_pos = ((double)(rand()) / RAND_MAX) * BOX_SIZE;
		atom_list[i].z_pos = ((double)(rand()) / RAND_MAX) * BOX_SIZE;
	}


	/* GPU memory allocation */
	hipMalloc((void**)&d_histogram_GPU, sizeof(bucket)*num_buckets);
	hipMalloc((void**)&d_atom_list, sizeof(atom)*total_num_data_points);
	
	/* GPU memory initialization */
	hipMemset(d_histogram_GPU, 0, sizeof(bucket)*num_buckets);
	hipMemcpy(d_atom_list, atom_list, sizeof(atom)*total_num_data_points, hipMemcpyHostToDevice);

	/* GPU block setting */
	int num_block;
	if( (total_num_data_points%32) != 0 ){
		num_block = (int)(total_num_data_points/32) + 1;
	} else {
		num_block = (int)total_num_data_points / BOX_SIZE;
	}
	dim3 grid(num_block, 1, 1);
	dim3 block(32, 1, 1);


	/******* time the GPU computing ***********************************/
	gettimeofday(&startTime, &Idunno);
	PDH_Kernel<<<grid, block>>>(d_histogram_GPU, d_atom_list, total_num_data_points, bucket_width);
	report_running_time_GPU();

	//Deal with memories
	hipMemcpy(histogram_GPU, d_histogram_GPU, sizeof(bucket)*num_buckets, hipMemcpyDeviceToHost);
	hipFree(d_histogram_GPU);
	hipFree(d_atom_list);

	//Output the result for GPU
	output_histogram_GPU();

	

	/******* time the CPU computing ***********************************/
	/* start counting time */
	gettimeofday(&startTime, &Idunno);
	
	/* call CPU single thread version to compute the histogram */
	PDH_baseline();
	
	/* check the total running time */ 
	report_running_time_CPU();
	
	/* print out the histogram */
	output_histogram();

	//Display the difference
	output_histogram_difference();
	
	return 0;
}


